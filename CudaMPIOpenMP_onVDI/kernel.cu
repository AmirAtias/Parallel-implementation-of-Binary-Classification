#include "hip/hip_runtime.h"

#include "training.h"
#ifndef __HIPCC__ 

#define __HIPCC__

#endif

#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>

#define BLOCKS_PER_BATCH 1000
#define MAX_SIZE_OF_THREADS  1000

// free allocated memory

hipError_t finalize(hipError_t cudaStatus, Point *d_points, float *d_resultArray, float *resultArray, float* d_weights, cudaVals* myCudaVals)

{
	hipFree(d_points);
	hipFree(d_resultArray);
	hipFree(d_weights);
	free(resultArray);
	myCudaVals->successAllocate = false;
	return cudaStatus;
}

hipError_t finalizeForMissPoints(hipError_t cudaStatus, Point* d_points, int * d_MissPoints, int * missPoints, float* d_weights, cudaVals* myCudaVals) {
	hipFree(d_points);
	hipFree(d_MissPoints);
	hipFree(d_weights);
	free(missPoints);
	myCudaVals->successAllocate = false;
	return cudaStatus;
}


__global__ void sumArray(Point * d_points, float * d_weights, float *d_resultArray) {

	extern __shared__ float s_point[];
	// load shared mem
	unsigned int tid = threadIdx.x;
	unsigned int bid = blockIdx.x;
	s_point[tid] = d_points[bid].coordinates[tid] * d_weights[tid];

	__syncthreads();
	// do reduction in shared mem
	for (unsigned int s = 1; s < blockDim.x; s *= 2) {
		if (tid % (2 * s) == 0) {
			s_point[tid] += s_point[tid + s];
		}
		__syncthreads();

	}

	if (tid == 0) {
		s_point[0] += d_weights[blockDim.x];
		d_resultArray[blockIdx.x] = s_point[0];

	}
}


hipError_t checkAllPointsLabel(int numOfPoints, int dimSize, Point *points, float *weights, cudaVals* myCudaVals) {
	int batchSize = numOfPoints / BLOCKS_PER_BATCH;
	int lastIter = numOfPoints%BLOCKS_PER_BATCH;
	Point retPoint, *d_points = 0;
	int findWrongPoint = 0;
	if (lastIter != 0)
		batchSize++;
	int counter = 0, i = 0;
	int blocksPerIter;
	float *d_weights = 0, *d_resultArray = 0, *resultArray = 0;
	hipError_t cudaStatus;
	cudaStatus = hipMalloc(&d_weights, (dimSize + 1) * sizeof(float));
	if (cudaStatus != hipSuccess) {
		printf("malloc d_weights failed!\n");
		return finalize(cudaStatus, d_points, d_resultArray, resultArray, d_weights, myCudaVals);
	}
	cudaStatus = hipMemcpy(d_weights, weights, (dimSize + 1) * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("hipMemcpy for d_weights failed!\n");
		return finalize(cudaStatus, d_points, d_resultArray, resultArray, d_weights, myCudaVals);
	}
	while (counter < batchSize && !findWrongPoint) {
		if (counter == batchSize - 1 && lastIter != 0) //last iteration
			blocksPerIter = lastIter;
		else
			blocksPerIter = BLOCKS_PER_BATCH;
		
		resultArray = (float *)malloc(blocksPerIter * sizeof(float));
		cudaStatus = hipMalloc(&d_resultArray, blocksPerIter * sizeof(float));
		if (cudaStatus != hipSuccess) {
			printf(" hipMalloc device result array failed!\n");
			return finalize(cudaStatus, d_points, d_resultArray, resultArray, d_weights, myCudaVals);
		}


		cudaStatus = hipMalloc((void**)&d_points, blocksPerIter * sizeof(Point));
		if (cudaStatus != hipSuccess) {
			printf("malloc d_points failed!\n");
			return finalize(cudaStatus, d_points, d_resultArray, resultArray, d_weights, myCudaVals);
		}
		cudaStatus = hipMemcpy(d_points, &points[counter*BLOCKS_PER_BATCH], blocksPerIter * sizeof(Point), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			printf("memcopy d_points failed!\n");
			return finalize(cudaStatus, d_points, d_resultArray, resultArray, d_weights, myCudaVals);
		}
		sumArray << <blocksPerIter, dimSize, dimSize * sizeof(float) >> > (d_points, d_weights, d_resultArray);//need to check dimsize-1
		cudaStatus = hipMemcpy(resultArray, d_resultArray, blocksPerIter * sizeof(float), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			printf("memcpy resultarray failed!\n");
			return finalize(cudaStatus, d_points, d_resultArray, resultArray, d_weights, myCudaVals);
		}
		for (i = 0; i < blocksPerIter; i++) {
			if (points[counter*BLOCKS_PER_BATCH + i].label == 1) {
				if (!(resultArray[i] > 0)) { //Miss
					retPoint = points[counter*BLOCKS_PER_BATCH + i];
					findWrongPoint = 1;
					break;
				}
			}
			else { //label==-1
				if (!(resultArray[i] < 0)) { //Miss
					retPoint = points[counter*BLOCKS_PER_BATCH + i];
					findWrongPoint = 1;
					break;
				}
			}
		}

		hipFree(d_points);
		hipFree(d_resultArray);
		free(resultArray);
		counter++;
	}
	hipFree(d_weights);
	if (findWrongPoint) {
		memcpy(&(myCudaVals->retPoint), &retPoint, sizeof(Point));
		return cudaStatus;

	}
	else {
		memcpy(&(myCudaVals->retPoint), &(points[numOfPoints - 1]), sizeof(Point));
		return cudaStatus;
	}
}
__global__ void calculateSumOfCoordinatesAllPoints(Point* d_points, int dimSize, int* d_MissPoints, float * d_weights, int startLocation) {
	int i, thread_index = threadIdx.x;
	int d_startLocation;
	if ((startLocation) == 0) {//case remainderEqualToZero || first function call
		int block_index = blockIdx.x;
		d_startLocation = block_index * blockDim.x;
	}
	else
		d_startLocation = startLocation;
	int index = thread_index + (d_startLocation); //(*d_startLocation);
	float sumCoordinates = 0;
	for (i = 0; i < dimSize; i++)
		sumCoordinates += d_points[index].coordinates[i] * d_weights[i];
	sumCoordinates += d_weights[i];//bias
	if (d_points[index].label == 1) {
		if ((sumCoordinates > 0))
			d_MissPoints[index] = 0;
		else
			d_MissPoints[index] = 1;
	}
	else { //label==-1
		if ((sumCoordinates < 0))
			d_MissPoints[index] = 0;
		else
			d_MissPoints[index] = 1;
	}
}
__global__ void calculateSumOfMissPoints(int *d_MissPoints, int sizeOfthreads)

{
	int index = threadIdx.x *sizeOfthreads, i;
	for (i = index + 1; i < index + sizeOfthreads; i++)
		if (d_MissPoints[i] != 0)
			d_MissPoints[index]++;
}

hipError_t numberOfMissPoints(int numOfPoints, int dimSize, Point *points, float *weights, cudaVals* myCudaVals) {
	float *d_weights = 0;
	hipError_t cudaStatus;
	Point *d_points = 0;
	int* d_MissPoints = 0, *missPoints = 0;
	int numberofBlocks, numberOfThreads, remainderEqualToZero, startLocation = 0;
	if (numOfPoints / MAX_SIZE_OF_THREADS > 0) {// case number of point bigger then max Num of threads
		numberofBlocks = numOfPoints / MAX_SIZE_OF_THREADS;
		numberOfThreads = MAX_SIZE_OF_THREADS;
	}
	else {// number of points < max number of threads
		numberofBlocks = 1;
		numberOfThreads = numOfPoints;
	}
	if (numOfPoints%MAX_SIZE_OF_THREADS != 0 && numOfPoints / MAX_SIZE_OF_THREADS > 0)// check if there is remainder
		remainderEqualToZero = 0;
	else
		remainderEqualToZero = 1;

	cudaStatus = hipMalloc(&d_weights, (dimSize + 1) * sizeof(float));
	if (cudaStatus != hipSuccess) {
		printf("hipMalloc for d_weights failed!\n");
		return finalizeForMissPoints(cudaStatus, d_points, d_MissPoints, missPoints, d_weights, myCudaVals);
	}
	cudaStatus = hipMemcpy(d_weights, weights, (dimSize + 1) * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("cuda memcpy for d_weights failed\n");
		return finalizeForMissPoints(cudaStatus, d_points, d_MissPoints, missPoints, d_weights, myCudaVals);
	}

	missPoints = (int *)malloc(numOfPoints * sizeof(int));
	cudaStatus = hipMalloc(&d_MissPoints, numOfPoints * sizeof(int));
	if (cudaStatus != hipSuccess) {
		printf("cuda malloc for d_MissPoints failed\n");
		return finalizeForMissPoints(cudaStatus, d_points, d_MissPoints, missPoints, d_weights, myCudaVals);
	}

	cudaStatus = hipMalloc((void**)&d_points, numOfPoints * sizeof(Point));
	if (cudaStatus != hipSuccess) {
		printf("malloc d_points failed\n");
		return finalizeForMissPoints(cudaStatus, d_points, d_MissPoints, missPoints, d_weights, myCudaVals);
	}
	cudaStatus = hipMemcpy(d_points, points, numOfPoints * sizeof(Point), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		printf("memcpy d_points failed\n");
		return finalizeForMissPoints(cudaStatus, d_points, d_MissPoints, missPoints, d_weights, myCudaVals);
	}

	calculateSumOfCoordinatesAllPoints << <numberofBlocks, numberOfThreads >> > (d_points, dimSize, d_MissPoints, d_weights, startLocation);

	if (!remainderEqualToZero) {
		numberOfThreads = numOfPoints%MAX_SIZE_OF_THREADS;
		startLocation = numOfPoints / MAX_SIZE_OF_THREADS;
		startLocation *= MAX_SIZE_OF_THREADS;// startLocation contain the start position of remainder
		calculateSumOfCoordinatesAllPoints << <1, numberOfThreads >> > (d_points, dimSize, d_MissPoints, d_weights, startLocation);
	}
	if (numberofBlocks > 0)
		calculateSumOfMissPoints << <1, numberofBlocks >> > (d_MissPoints, numberOfThreads);
	cudaStatus = hipMemcpy(missPoints, d_MissPoints, numOfPoints * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		printf("memcpy d_MissPoints failed\n");
		return finalizeForMissPoints(cudaStatus, d_points, d_MissPoints, missPoints, d_weights, myCudaVals);
	}
	int retVal = 0, i;
	for (i = 0; i < numOfPoints; i += numberOfThreads) {
		retVal += missPoints[i];
	}
	if (!remainderEqualToZero) { //case there is remainder -need to calculate separately
		for (i = startLocation; i < numOfPoints; i++) {// startLocation contain the start position of remainder
			retVal += missPoints[i];
		}
	}
	free(missPoints);
	hipFree(d_weights);
	hipFree(d_points);
	hipFree(d_MissPoints);
	myCudaVals->numOfMissPoint = retVal;
	return cudaStatus;
}




